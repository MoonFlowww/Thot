#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <iostream>
#include "../../cuh/layers/conv2d.cuh"

#ifdef THOT_WITH_CUDNN
#include <hipDNN.h>
#endif

namespace cuda {
    namespace layers {

        // Transform input into column matrix
        __global__ void im2col_kernel(const float* input, float* col,
            int batch_size, int channels, int height, int width,
            int kernel_size, int stride, int padding,
            int out_height, int out_width) {
            int K = channels * kernel_size * kernel_size;
            int N = batch_size * out_height * out_width;
            int idx = blockIdx.x * blockDim.x + threadIdx.x;
            if (idx >= K * N) return;

            int n = idx / K;
            int k = idx % K;
            int b = n / (out_height * out_width);
            int oh = (n / out_width) % out_height;
            int ow = n % out_width;

            int c = k / (kernel_size * kernel_size);
            int kh = (k / kernel_size) % kernel_size;
            int kw = k % kernel_size;

            int ih = oh * stride - padding + kh;
            int iw = ow * stride - padding + kw;
            float val = 0.0f;
            if (ih >= 0 && ih < height && iw >= 0 && iw < width) {
                int in_idx = b * (channels * height * width) +
                    c * (height * width) +
                    ih * width +
                    iw;
                val = input[in_idx];
            }
            col[k * N + n] = val;
        }

        __global__ void add_bias(float* C, const float* bias, int M, int N) {
            int idx = blockIdx.x * blockDim.x + threadIdx.x;
            if (idx >= M * N) return;
            int oc = idx / N;
            C[idx] += bias[oc];
        }

        void launchConv2DForwardIm2Col(const float* input, const float* weights, const float* bias,
            float* output, int batch_size, int in_channels, int in_height, int in_width,
            int out_channels, int kernel_size, int stride, int padding, int out_height, int out_width,
            hipStream_t stream) {

            int K = in_channels * kernel_size * kernel_size;
            int N = batch_size * out_height * out_width;
            size_t col_size = static_cast<size_t>(K) * N * sizeof(float);
            float* col;
            hipMalloc(&col, col_size);

            int threads = 256;
            int blocks = (K * N + threads - 1) / threads;
            im2col_kernel<<<blocks, threads, 0, stream>>>(input, col,
                batch_size, in_channels, in_height, in_width,
                kernel_size, stride, padding,
                out_height, out_width);

            hipblasHandle_t handle;
            hipblasCreate(&handle);
            hipblasSetStream(handle, stream);

            const float alpha = 1.0f;
            const float beta = 0.0f;
            // weights: [out_channels, K], col: [K, N], output: [out_channels, N]
            hipblasStatus_t stat = hipblasSgemm(handle,
                HIPBLAS_OP_N, HIPBLAS_OP_N,
                N, out_channels, K,
                &alpha,
                col, N,
                weights, K,
                &beta,
                output, N);
            if (stat != HIPBLAS_STATUS_SUCCESS) {
                printf("cuBLAS sgemm failed in launchConv2DForwardIm2Col\n");
            }

            if (bias != nullptr) {
                int total = out_channels * N;
                int blockB = 256;
                int gridB = (total + blockB - 1) / blockB;
                add_bias<<<gridB, blockB, 0, stream>>>(output, bias, out_channels, N);
            }

            hipblasDestroy(handle);
            hipFree(col);
            hipDeviceSynchronize();
        }

#ifdef THOT_WITH_CUDNN
        void launchConv2DForwardCuDNN(const float* input, const float* weights, const float* bias,
            float* output, int batch_size, int in_channels, int in_height, int in_width,
            int out_channels, int kernel_size, int stride, int padding, int out_height, int out_width,
            hipStream_t stream) {

            hipdnnHandle_t handle;
            hipdnnCreate(&handle);
            hipdnnSetStream(handle, stream);

            hipdnnTensorDescriptor_t in_desc, out_desc, bias_desc;
            hipdnnFilterDescriptor_t w_desc;
            hipdnnConvolutionDescriptor_t conv_desc;
            hipdnnCreateTensorDescriptor(&in_desc);
            hipdnnCreateTensorDescriptor(&out_desc);
            hipdnnCreateTensorDescriptor(&bias_desc);
            hipdnnCreateFilterDescriptor(&w_desc);
            hipdnnCreateConvolutionDescriptor(&conv_desc);

            hipdnnSetTensor4dDescriptor(in_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
                batch_size, in_channels, in_height, in_width);
            hipdnnSetTensor4dDescriptor(out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
                batch_size, out_channels, out_height, out_width);
            hipdnnSetTensor4dDescriptor(bias_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
                1, out_channels, 1, 1);
            hipdnnSetFilter4dDescriptor(w_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW,
                out_channels, in_channels, kernel_size, kernel_size);
            hipdnnSetConvolution2dDescriptor(conv_desc, padding, padding, stride, stride,
                1, 1, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT);

            hipdnnConvolutionFwdAlgo_t algo;
            hipdnnGetConvolutionForwardAlgorithm(handle, in_desc, w_desc, conv_desc, out_desc,
                HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &algo);

            size_t ws_size = 0;
            hipdnnGetConvolutionForwardWorkspaceSize(handle, in_desc, w_desc, conv_desc, out_desc,
                algo, &ws_size);
            void* workspace = nullptr;
            if (ws_size > 0) hipMalloc(&workspace, ws_size);

            const float alpha = 1.0f;
            const float beta = 0.0f;
            hipdnnConvolutionForward(handle, &alpha,
                in_desc, input,
                w_desc, weights,
                conv_desc, algo,
                workspace, ws_size,
                &beta,
                out_desc, output);

            if (bias != nullptr) {
                hipdnnAddTensor(handle, &alpha, bias_desc, bias, &alpha, out_desc, output);
            }

            if (workspace) hipFree(workspace);
            hipdnnDestroyTensorDescriptor(in_desc);
            hipdnnDestroyTensorDescriptor(out_desc);
            hipdnnDestroyTensorDescriptor(bias_desc);
            hipdnnDestroyFilterDescriptor(w_desc);
            hipdnnDestroyConvolutionDescriptor(conv_desc);
            hipdnnDestroy(handle);
            hipDeviceSynchronize();
        }

        void launchConv2DBackwardInputCuDNN(const float* grad_output, const float* weights,
            float* grad_input, int batch_size, int in_channels, int in_height, int in_width,
            int out_channels, int kernel_size, int stride, int padding, int out_height, int out_width,
            hipStream_t stream) {

            hipdnnHandle_t handle;
            hipdnnCreate(&handle);
            hipdnnSetStream(handle, stream);

            hipdnnTensorDescriptor_t grad_out_desc, grad_in_desc;
            hipdnnFilterDescriptor_t w_desc;
            hipdnnConvolutionDescriptor_t conv_desc;

            hipdnnCreateTensorDescriptor(&grad_out_desc);
            hipdnnCreateTensorDescriptor(&grad_in_desc);
            hipdnnCreateFilterDescriptor(&w_desc);
            hipdnnCreateConvolutionDescriptor(&conv_desc);

            hipdnnSetTensor4dDescriptor(grad_out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
                batch_size, out_channels, out_height, out_width);
            hipdnnSetTensor4dDescriptor(grad_in_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
                batch_size, in_channels, in_height, in_width);
            hipdnnSetFilter4dDescriptor(w_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW,
                out_channels, in_channels, kernel_size, kernel_size);
            hipdnnSetConvolution2dDescriptor(conv_desc, padding, padding, stride, stride,
                1, 1, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT);

            hipdnnConvolutionBwdDataAlgo_t algo;
            hipdnnGetConvolutionBackwardDataAlgorithm(handle, w_desc, grad_out_desc, conv_desc, grad_in_desc,
                HIPDNN_CONVOLUTION_BWD_DATA_PREFER_FASTEST, 0, &algo);
            size_t ws_size = 0;
            hipdnnGetConvolutionBackwardDataWorkspaceSize(handle, w_desc, grad_out_desc, conv_desc, grad_in_desc,
                algo, &ws_size);
            void* workspace = nullptr;
            if (ws_size > 0) hipMalloc(&workspace, ws_size);

            const float alpha = 1.0f;
            const float beta = 0.0f;
            hipdnnConvolutionBackwardData(handle, &alpha,
                w_desc, weights,
                grad_out_desc, grad_output,
                conv_desc, algo,
                workspace, ws_size,
                &beta,
                grad_in_desc, grad_input);

            if (workspace) hipFree(workspace);
            hipdnnDestroyTensorDescriptor(grad_out_desc);
            hipdnnDestroyTensorDescriptor(grad_in_desc);
            hipdnnDestroyFilterDescriptor(w_desc);
            hipdnnDestroyConvolutionDescriptor(conv_desc);
            hipdnnDestroy(handle);
            hipDeviceSynchronize();
        }

        void launchConv2DBackwardWeightsCuDNN(const float* input, const float* grad_output,
            float* grad_weights, int batch_size, int in_channels, int in_height, int in_width,
            int out_channels, int kernel_size, int stride, int padding, int out_height, int out_width,
            hipStream_t stream) {

            hipdnnHandle_t handle;
            hipdnnCreate(&handle);
            hipdnnSetStream(handle, stream);

            hipdnnTensorDescriptor_t grad_out_desc, in_desc;
            hipdnnFilterDescriptor_t grad_w_desc;
            hipdnnConvolutionDescriptor_t conv_desc;

            hipdnnCreateTensorDescriptor(&grad_out_desc);
            hipdnnCreateTensorDescriptor(&in_desc);
            hipdnnCreateFilterDescriptor(&grad_w_desc);
            hipdnnCreateConvolutionDescriptor(&conv_desc);

            hipdnnSetTensor4dDescriptor(grad_out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
                batch_size, out_channels, out_height, out_width);
            hipdnnSetTensor4dDescriptor(in_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
                batch_size, in_channels, in_height, in_width);
            hipdnnSetFilter4dDescriptor(grad_w_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW,
                out_channels, in_channels, kernel_size, kernel_size);
            hipdnnSetConvolution2dDescriptor(conv_desc, padding, padding, stride, stride,
                1, 1, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT);

            hipdnnConvolutionBwdFilterAlgo_t algo;
            hipdnnGetConvolutionBackwardFilterAlgorithm(handle, in_desc, grad_out_desc, conv_desc, grad_w_desc,
                HIPDNN_CONVOLUTION_BWD_FILTER_PREFER_FASTEST, 0, &algo);
            size_t ws_size = 0;
            hipdnnGetConvolutionBackwardFilterWorkspaceSize(handle, in_desc, grad_out_desc, conv_desc, grad_w_desc,
                algo, &ws_size);
            void* workspace = nullptr;
            if (ws_size > 0) hipMalloc(&workspace, ws_size);

            const float alpha = 1.0f;
            const float beta = 0.0f;
            hipdnnConvolutionBackwardFilter(handle, &alpha,
                in_desc, input,
                grad_out_desc, grad_output,
                conv_desc, algo,
                workspace, ws_size,
                &beta,
                grad_w_desc, grad_weights);

            if (workspace) hipFree(workspace);
            hipdnnDestroyTensorDescriptor(grad_out_desc);
            hipdnnDestroyTensorDescriptor(in_desc);
            hipdnnDestroyFilterDescriptor(grad_w_desc);
            hipdnnDestroyConvolutionDescriptor(conv_desc);
            hipdnnDestroy(handle);
            hipDeviceSynchronize();
        }

        void launchConv2DBackwardBiasCuDNN(const float* grad_output, float* grad_bias,
            int batch_size, int out_channels, int out_height, int out_width,
            hipStream_t stream) {

            hipdnnHandle_t handle;
            hipdnnCreate(&handle);
            hipdnnSetStream(handle, stream);

            hipdnnTensorDescriptor_t grad_out_desc, bias_desc;
            hipdnnCreateTensorDescriptor(&grad_out_desc);
            hipdnnCreateTensorDescriptor(&bias_desc);

            hipdnnSetTensor4dDescriptor(grad_out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
                batch_size, out_channels, out_height, out_width);
            hipdnnSetTensor4dDescriptor(bias_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
                1, out_channels, 1, 1);

            const float alpha = 1.0f;
            const float beta = 0.0f;
            hipdnnConvolutionBackwardBias(handle, &alpha,
                grad_out_desc, grad_output,
                &beta,
                bias_desc, grad_bias);

            hipdnnDestroyTensorDescriptor(grad_out_desc);
            hipdnnDestroyTensorDescriptor(bias_desc);
            hipdnnDestroy(handle);
            hipDeviceSynchronize();
        }
#endif // THOT_WITH_CUDNN

    }
}
