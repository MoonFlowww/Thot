#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>
#include <thrust/reduce.h>

#include "../../cuh/losses/loss.cuh"

namespace cuda {
    namespace losses {


        __device__ bool verbose = false;


        // Mean Squared Error (MSE)
        __global__ void mse(const float* predictions, const float* targets, float* loss, int size) {
            int idx = blockIdx.x * blockDim.x + threadIdx.x;
            if (idx < size) {
                float diff = predictions[idx] - targets[idx];
                loss[idx] = 0.5f * diff * diff;
                if (verbose && loss[idx]!=0) printf("MSE loss[%d] = %f\n", idx, loss[idx]);
            }
        }

        __global__ void mseGradient(const float* predictions, const float* targets, float* gradients, int size) {
            int idx = blockIdx.x * blockDim.x + threadIdx.x;
            if (idx < size) {
                gradients[idx] = predictions[idx] - targets[idx];
                if (verbose && gradients[idx]!=0) printf("MSE grad[%d] = %f\n", idx, gradients[idx]);
            }
        }

        // Mean Absolute Error (MAE)
        __global__ void mae(const float* predictions, const float* targets, float* loss, int size) {
            int idx = blockIdx.x * blockDim.x + threadIdx.x;
            if (idx < size) {
                loss[idx] = fabsf(predictions[idx] - targets[idx]);
                if (verbose && loss[idx]!=0) printf("MAE loss[%d] = %f\n", idx, loss[idx]);
            }
        }

        __global__ void maeGradient(const float* predictions, const float* targets, float* gradients, int size) {
            int idx = blockIdx.x * blockDim.x + threadIdx.x;
            if (idx < size) {
                float diff = predictions[idx] - targets[idx];
                gradients[idx] = (diff > 0.0f) ? 1.0f : ((diff < 0.0f) ? -1.0f : 0.0f);
                if (verbose && gradients[idx]!=0) printf("MAE grad[%d] = %f\n", idx, gradients[idx]);
            }
        }

        // Binary Cross-Entropy
        __global__ void binaryCrossEntropy(const float* predictions, const float* targets, float* loss, int size, float epsilon) {
            int idx = blockIdx.x * blockDim.x + threadIdx.x;
            if (idx < size) {
                float p = fmaxf(fminf(predictions[idx], 1.0f - epsilon), epsilon);
                float t = targets[idx];
                loss[idx] = -(t * logf(p) + (1.0f - t) * logf(1.0f - p));
                if (verbose && loss[idx]!=0) printf("BCE loss[%d] = %f\n", idx, loss[idx]);
            }
        }

        __global__ void binaryCrossEntropyGradient(const float* predictions, const float* targets, float* gradients, int size, float epsilon) {
            int idx = blockIdx.x * blockDim.x + threadIdx.x;
            if (idx < size) {
                float p = fmaxf(fminf(predictions[idx], 1.0f - epsilon), epsilon);
                float t = targets[idx];
                gradients[idx] = -t / p + (1.0f - t) / (1.0f - p);
                if (verbose && gradients[idx]!=0) printf("BCE grad[%d] = %f\n", idx, gradients[idx]);
            }
        }


        __global__ void crossEntropy(const float* predictions, const float* targets, float* loss, int size, float epsilon) {
            int idx = blockIdx.x * blockDim.x + threadIdx.x;
            if (idx < size) {
                float p = fmaxf(predictions[idx], epsilon);
                float t = targets[idx];
                loss[idx] = -t * logf(p);
                if (verbose && loss[idx]!=0) printf("CE loss[%d] = %f\n", idx, loss[idx]);
            }
        }

        __global__ void crossEntropyGradient(const float* predictions, const float* targets, float* gradients, int size, float epsilon) {
            int idx = blockIdx.x * blockDim.x + threadIdx.x;
            if (idx < size) {
                float p = fmaxf(predictions[idx], epsilon);
                float t = targets[idx];
                gradients[idx] = -t / p;
                if (verbose && gradients[idx]!=0) printf("CE grad[%d] = %f\n", idx, gradients[idx]);
            }
        }



        // Categorical Cross-Entropy
        __global__ void categoricalCrossEntropy(const float* predictions,
                                                const float* targets,
                                                float* loss,
                                                int num_classes,
                                                float epsilon) {
            extern __shared__ float sdata[];
            int b = blockIdx.x;
            int tid = threadIdx.x;
            const float* p_row = predictions + b * num_classes;
            const float* t_row = targets + b * num_classes;

            float local = 0.0f;
            int vec_stride = blockDim.x * 4;
            int limit = num_classes & ~3;
            for (int i = tid * 4; i < limit; i += vec_stride) {
                float4 p = reinterpret_cast<const float4*>(p_row)[i / 4];
                float4 t = reinterpret_cast<const float4*>(t_row)[i / 4];
                local += -t.x * logf(fmaxf(p.x, epsilon))
                       + -t.y * logf(fmaxf(p.y, epsilon))
                       + -t.z * logf(fmaxf(p.z, epsilon))
                       + -t.w * logf(fmaxf(p.w, epsilon));
            }
            for (int i = limit + tid; i < num_classes; i += blockDim.x) {
                float p = fmaxf(p_row[i], epsilon);
                float t = t_row[i];
                local += -t * logf(p);
            }

            sdata[tid] = local;
            __syncthreads();
            for (int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
                if (tid < offset)
                    sdata[tid] += sdata[tid + offset];
                __syncthreads();
            }
            if (tid == 0) {
                loss[b] = sdata[0];
                if (verbose && loss[b] != 0) printf("CCE loss[%d] = %f\n", b, loss[b]);
            }
        }

        __global__ void categoricalCrossEntropyGradient(const float* predictions,
                                                        const float* targets,
                                                        float* gradients,
                                                        int num_classes,
                                                        float epsilon) {
            int b = blockIdx.x;
            int tid = threadIdx.x;
            const float* p_row = predictions + b * num_classes;
            const float* t_row = targets + b * num_classes;
            float* g_row = gradients + b * num_classes;

            int vec_stride = blockDim.x * 4;
            int limit = num_classes & ~3;
            for (int i = tid * 4; i < limit; i += vec_stride) {
                float4 p = reinterpret_cast<const float4*>(p_row)[i / 4];
                float4 t = reinterpret_cast<const float4*>(t_row)[i / 4];
                float4 g;
                g.x = -t.x / fmaxf(p.x, epsilon);
                g.y = -t.y / fmaxf(p.y, epsilon);
                g.z = -t.z / fmaxf(p.z, epsilon);
                g.w = -t.w / fmaxf(p.w, epsilon);
                reinterpret_cast<float4*>(g_row)[i / 4] = g;
            }
            for (int i = limit + tid; i < num_classes; i += blockDim.x) {
                float p = fmaxf(p_row[i], epsilon);
                float t = t_row[i];
                g_row[i] = -t / p;
            }
            if (verbose && tid == 0)
                for (int i = 0; i < num_classes; ++i)
                    if (g_row[i] != 0)
                        printf("CCE grad[%d] = %f\n", b * num_classes + i, g_row[i]);
        }


        // Sparse Categorical Cross-Entropy
        __global__ void sparseCategoricalCrossEntropy(const float* predictions, const float* targets, float* loss, int batch_size, int num_classes, float epsilon) {
            int idx = blockIdx.x * blockDim.x + threadIdx.x;
            if (idx < batch_size) {
                int target_class = targets[idx];
                if (target_class >= 0 && target_class < num_classes) {
                    float p = fmaxf(predictions[idx * num_classes + target_class], epsilon);
                    loss[idx] = -logf(p);
                } else {
                    loss[idx] = 0.0f;
                }
                if (verbose && loss[idx]!=0) printf("Sparse CCE loss[%d] = %f\n", idx, loss[idx]);
            }
        }

        __global__ void sparseCategoricalCrossEntropyGradient(const float* predictions,
                                                              const float* targets,
                                                              float* gradients,
                                                              int num_classes,
                                                              float epsilon) {
            int b = blockIdx.x;
            int target = static_cast<int>(targets[b]);
            const float* p_row = predictions + b * num_classes;
            float* g_row = gradients + b * num_classes;
            int tid = threadIdx.x;
            for (int i = tid; i < num_classes; i += blockDim.x) {
                if (i == target)
                    g_row[i] = -1.0f / fmaxf(p_row[i], epsilon);
                else
                    g_row[i] = 0.0f;
            }
            if (verbose && tid == 0 && target >= 0 && target < num_classes)
                printf("Sparse CCE grad[%d] = %f\n", b * num_classes + target,
                       g_row[target]);
        }

        // Hinge
        __global__ void hinge(const float* predictions, const float* targets, float* loss, int size) {
            int idx = blockIdx.x * blockDim.x + threadIdx.x;
            if (idx < size) {
                float margin = 1.0f - predictions[idx] * targets[idx];
                loss[idx] = fmaxf(0.0f, margin);
                if (verbose && loss[idx]!=0) printf("Hinge loss[%d] = %f\n", idx, loss[idx]);
            }
        }

        __global__ void hingeGradient(const float* predictions, const float* targets, float* gradients, int size) {
            int idx = blockIdx.x * blockDim.x + threadIdx.x;
            if (idx < size) {
                float margin = 1.0f - predictions[idx] * targets[idx];
                gradients[idx] = (margin > 0.0f) ? -targets[idx] : 0.0f;
                if (verbose && gradients[idx]!=0) printf("Hinge grad[%d] = %f\n", idx, gradients[idx]);
            }
        }

        // Huber
        __global__ void huber(const float* predictions, const float* targets, float* loss, int size, float delta) {
            int idx = blockIdx.x * blockDim.x + threadIdx.x;
            if (idx < size) {
                float diff = fabsf(predictions[idx] - targets[idx]);
                loss[idx] = (diff <= delta) ?
                    0.5f * diff * diff :
                    delta * (diff - 0.5f * delta);
                if (verbose && loss[idx]!=0) printf("Huber loss[%d] = %f\n", idx, loss[idx]);
            }
        }

        __global__ void huberGradient(const float* predictions, const float* targets, float* gradients, int size, float delta) {
            int idx = blockIdx.x * blockDim.x + threadIdx.x;
            if (idx < size) {
                float diff = predictions[idx] - targets[idx];
                gradients[idx] = (fabsf(diff) <= delta) ?
                    diff :
                    delta * ((diff > 0.0f) ? 1.0f : -1.0f);
                if (verbose && gradients[idx]!=0) printf("Huber grad[%d] = %f\n", idx, gradients[idx]);
            }
        }

        // KL Divergence
        __global__ void klDivergence(const float* predictions, const float* targets, float* loss, int size, float epsilon) {
            int idx = blockIdx.x * blockDim.x + threadIdx.x;
            if (idx < size) {
                float p = fmaxf(predictions[idx], epsilon);
                float q = fmaxf(targets[idx], epsilon);
                loss[idx] = q * logf(q / p);
                if (verbose && loss[idx]!=0) printf("KL loss[%d] = %f\n", idx, loss[idx]);
            }
        }

        __global__ void klDivergenceGradient(const float* predictions, const float* targets, float* gradients, int size, float epsilon) {
            int idx = blockIdx.x * blockDim.x + threadIdx.x;
            if (idx < size) {
                float p = fmaxf(predictions[idx], epsilon);
                float q = fmaxf(targets[idx], epsilon);
                gradients[idx] = -q / p;
                if (verbose && gradients[idx]!=0) printf("KL grad[%d] = %f\n", idx, gradients[idx]);
            }
        }

        // Wrapper functions for launching 
        void launchMSE(const float* predictions, const float* targets, float* loss, int size, hipStream_t stream) {
            int blockSize = 256;
            int numBlocks = (size + blockSize - 1) / blockSize;
            mse << <numBlocks, blockSize, 0, stream >> > (predictions, targets, loss, size);
            hipError_t err = hipGetLastError();
            if (err != hipSuccess) printf("Kernel launch error in launchMSE: %s\n", hipGetErrorString(err));
            hipDeviceSynchronize();
        }

        void launchMSEGradient(const float* predictions, const float* targets, float* gradients, int size, hipStream_t stream) {
            int blockSize = 256;
            int numBlocks = (size + blockSize - 1) / blockSize;
            mseGradient << <numBlocks, blockSize, 0, stream >> > (predictions, targets, gradients, size);
            hipError_t err = hipGetLastError();
            if (err != hipSuccess) printf("Kernel launch error in launchMSEGradient: %s\n", hipGetErrorString(err));
            hipDeviceSynchronize();
        }

        void launchMAE(const float* predictions, const float* targets, float* loss, int size, hipStream_t stream) {
            int blockSize = 256;
            int numBlocks = (size + blockSize - 1) / blockSize;
            mae << <numBlocks, blockSize, 0, stream >> > (predictions, targets, loss, size);
            hipError_t err = hipGetLastError();
            if (err != hipSuccess) printf("Kernel launch error in launchMAE: %s\n", hipGetErrorString(err));
            hipDeviceSynchronize();
        }

        void launchMAEGradient(const float* predictions, const float* targets, float* gradients, int size, hipStream_t stream) {
            int blockSize = 256;
            int numBlocks = (size + blockSize - 1) / blockSize;
            maeGradient << <numBlocks, blockSize, 0, stream >> > (predictions, targets, gradients, size);
            hipError_t err = hipGetLastError();
            if (err != hipSuccess) printf("Kernel launch error in launchMAEGradient: %s\n", hipGetErrorString(err));
            hipDeviceSynchronize();
        }

        void launchBinaryCrossEntropy(const float* predictions, const float* targets, float* loss, int size, float epsilon, hipStream_t stream) {
            int blockSize = 256;
            int numBlocks = (size + blockSize - 1) / blockSize;
            binaryCrossEntropy << <numBlocks, blockSize, 0, stream >> > (predictions, targets, loss, size, epsilon);
            hipError_t err = hipGetLastError();
            if (err != hipSuccess) printf("Kernel launch error in launchBinaryCrossEntropy: %s\n", hipGetErrorString(err));
            hipDeviceSynchronize();
        }

        void launchBinaryCrossEntropyGradient(const float* predictions, const float* targets, float* gradients, int size, float epsilon, hipStream_t stream) {
            int blockSize = 256;
            int numBlocks = (size + blockSize - 1) / blockSize;
            binaryCrossEntropyGradient << <numBlocks, blockSize, 0, stream >> > (predictions, targets, gradients, size, epsilon);
            hipError_t err = hipGetLastError();
            if (err != hipSuccess) printf("Kernel launch error in launchBinaryCrossEntropyGradient: %s\n", hipGetErrorString(err));
            hipDeviceSynchronize();
        }

        void launchCrossEntropy(const float* predictions, const float* targets, float* loss, int size, float epsilon, hipStream_t stream) {
            int blockSize = 256;
            int numBlocks = (size + blockSize - 1) / blockSize;
            crossEntropy<<<numBlocks, blockSize, 0, stream>>>(predictions, targets, loss, size, epsilon);
            hipError_t err = hipGetLastError();
            if (err != hipSuccess) printf("Kernel launch error in launchCrossEntropy: %s\n", hipGetErrorString(err));
            hipDeviceSynchronize();
        }

        void launchCrossEntropyGradient(const float* predictions, const float* targets, float* gradients, int size, float epsilon, hipStream_t stream) {
            int blockSize = 256;
            int numBlocks = (size + blockSize - 1) / blockSize;
            crossEntropyGradient<<<numBlocks, blockSize, 0, stream>>>(predictions, targets, gradients, size, epsilon);
            hipError_t err = hipGetLastError();
            if (err != hipSuccess) printf("Kernel launch error in launchCrossEntropyGradient: %s\n", hipGetErrorString(err));
            hipDeviceSynchronize();
        }

        void launchCategoricalCrossEntropy(const float* predictions, const float* targets, float* loss, int batch_size, int num_classes, float epsilon, hipStream_t stream) {
            int blockSize = 256;
            size_t shared = blockSize * sizeof(float);
            categoricalCrossEntropy<<<batch_size, blockSize, shared, stream>>>( predictions, targets, loss, num_classes, epsilon);
            hipError_t err = hipGetLastError();
            if (err != hipSuccess)
                printf("Kernel launch error in launchCategoricalCrossEntropy: %s\n", hipGetErrorString(err));
            hipDeviceSynchronize();
        }

        void launchCategoricalCrossEntropyGradient(const float* predictions, const float* targets, float* gradients, int batch_size, int num_classes, float epsilon, hipStream_t stream) {
            int blockSize = 256;
            categoricalCrossEntropyGradient<<<batch_size, blockSize, 0, stream>>>(predictions, targets, gradients, num_classes, epsilon);
            hipError_t err = hipGetLastError();
            if (err != hipSuccess)
                printf("Kernel launch error in launchCategoricalCrossEntropyGradient: %s\n", hipGetErrorString(err));
            hipDeviceSynchronize();
        }

        void launchSparseCategoricalCrossEntropy(const float* predictions, const float* targets, float* loss, int batch_size, int num_classes, float epsilon, hipStream_t stream) {
            int blockSize = 256;
            int numBlocks = (batch_size + blockSize - 1) / blockSize;
            sparseCategoricalCrossEntropy<<<numBlocks, blockSize, 0, stream>>>(predictions, targets, loss, batch_size, num_classes, epsilon);
            hipError_t err = hipGetLastError();
            if (err != hipSuccess)
                printf("Kernel launch error in launchSparseCategoricalCrossEntropy: %s\n", hipGetErrorString(err));
            hipDeviceSynchronize();
        }

        void launchSparseCategoricalCrossEntropyGradient(const float* predictions, const float* targets, float* gradients, int batch_size, int num_classes, float epsilon, hipStream_t stream) {
            int blockSize = 256;
            sparseCategoricalCrossEntropyGradient<<<batch_size, blockSize, 0, stream>>>(predictions, targets, gradients, num_classes, epsilon);
            hipError_t err = hipGetLastError();
            if (err != hipSuccess)
                printf("Kernel launch error in launchSparseCategoricalCrossEntropyGradient: %s\n", hipGetErrorString(err));
            hipDeviceSynchronize();
        }

        void launchHinge(const float* predictions, const float* targets, float* loss, int size, hipStream_t stream) {
            int blockSize = 256;
            int numBlocks = (size + blockSize - 1) / blockSize;
            hinge << <numBlocks, blockSize, 0, stream >> > (predictions, targets, loss, size);
            hipError_t err = hipGetLastError();
            if (err != hipSuccess) printf("Kernel launch error in launchHinge: %s\n", hipGetErrorString(err));
            hipDeviceSynchronize();
        }

        void launchHingeGradient(const float* predictions, const float* targets, float* gradients, int size, hipStream_t stream) {
            int blockSize = 256;
            int numBlocks = (size + blockSize - 1) / blockSize;
            hingeGradient << <numBlocks, blockSize, 0, stream >> > (predictions, targets, gradients, size);
            hipError_t err = hipGetLastError();
            if (err != hipSuccess) printf("Kernel launch error in launchHingeGradient: %s\n", hipGetErrorString(err));
            hipDeviceSynchronize();
        }

        void launchHuber(const float* predictions, const float* targets, float* loss, int size, float delta, hipStream_t stream) {
            int blockSize = 256;
            int numBlocks = (size + blockSize - 1) / blockSize;
            huber << <numBlocks, blockSize, 0, stream >> > (predictions, targets, loss, size, delta);
            hipError_t err = hipGetLastError();
            if (err != hipSuccess) printf("Kernel launch error in launchHuber: %s\n", hipGetErrorString(err));
            hipDeviceSynchronize();
        }

        void launchHuberGradient(const float* predictions, const float* targets, float* gradients, int size, float delta, hipStream_t stream) {
            int blockSize = 256;
            int numBlocks = (size + blockSize - 1) / blockSize;
            huberGradient << <numBlocks, blockSize, 0, stream >> > (predictions, targets, gradients, size, delta);
            hipError_t err = hipGetLastError();
            if (err != hipSuccess) printf("Kernel launch error in launchHuberGradient: %s\n", hipGetErrorString(err));
            hipDeviceSynchronize();
        }

        void launchKLDivergence(const float* predictions, const float* targets, float* loss, int size, float epsilon, hipStream_t stream) {
            int blockSize = 256;
            int numBlocks = (size + blockSize - 1) / blockSize;
            klDivergence << <numBlocks, blockSize, 0, stream >> > (predictions, targets, loss, size, epsilon);
            hipError_t err = hipGetLastError();
            if (err != hipSuccess) printf("Kernel launch error in launchKLDivergence: %s\n", hipGetErrorString(err));
            hipDeviceSynchronize();
        }

        void launchKLDivergenceGradient(const float* predictions, const float* targets, float* gradients, int size, float epsilon, hipStream_t stream) {
            int blockSize = 256;
            int numBlocks = (size + blockSize - 1) / blockSize;
            klDivergenceGradient << <numBlocks, blockSize, 0, stream >> > (predictions, targets, gradients, size, epsilon);
            hipError_t err = hipGetLastError();
            if (err != hipSuccess) printf("Kernel launch error in launchKLDivergenceGradient: %s\n", hipGetErrorString(err));
            hipDeviceSynchronize();
        }


        float reduceLoss(float* loss, int size, hipStream_t stream) {
            thrust::device_ptr<float> loss_ptr(loss);
            return thrust::reduce(thrust::cuda::par.on(stream), loss_ptr, loss_ptr + size, 0.0f);
        }


    }  // namespace losses

}  // namespace cuda 